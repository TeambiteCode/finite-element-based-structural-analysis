#define _USE_MATH_DEFINES

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

void print_arr(int*, int, int);
void print_arr(double*, int, int);
double determinant(double* A, int n);
void adjoint(double* A, double* adj, int n);
bool inverse(double* A, double* inverse, int n);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
	//Nodal coordinates
	//-----------------

	const int x[] = { 0, 6, 12, 18, 24, 18, 12, 6 };
	const int y[] = { 0, 0, 0,  0,  0,  6,  6,  6 };

	//Element connectivity(ECM)
	// -------------------------

	const int ECM[2][13] = { {0, 1, 2, 3, 4, 5, 6, 0, 1, 1, 2, 3, 3},
						  {1, 2, 3, 4, 5, 6, 7, 7, 7, 6, 6, 6, 5} };

	//Material properites
	//--------------------

	const long	E = 200 * (1000000);

	//Geomatric properties
	//---------------------

	const double CArea[] = { .0012, .0012, .0012, .0012, .0012, .0012, .0012, .0012, .0012, .0012, .0012, .0012, .0012 }; //cross section area
	const double	L[] = { 6,     6,     6,     6,  6 * sqrt(2), 6,    6,  6 * sqrt(2), 6, 6 * sqrt(2), 6,  6 * sqrt(2), 6 };

	//Additional input parameters required for coding
	//-----------------------------------------------

	const int	 nn = 8;	//number of nodes
	const int	nel = 13;   //number of el ements
	const int	nen = 2;	//number of nodes in an element
	const int  ndof = 2;	//number of dof per node

	int	tdof = nn*ndof;
	double Ang[] = { 0, 0, 0, 0, M_PI / 4 * 3, M_PI, M_PI, M_PI / 4 , M_PI / 2, M_PI / 4, M_PI / 2, M_PI / 4 * 3, M_PI / 2 };
	int LCM[2 * ndof][nel] = {}; //Local Coo matrix
	double K[nn*ndof][nn*ndof] = {}; //Global stiffness matrix

	//Restrained DOF to Apply BC
	//---------------------------

	int BC[] = { 1, 2, 10 }; //restrained dof

	//Local coordinate matrix(LCM)
	// -----------------------------
	int ind;
	for (int j = 0; j < nen; j++) {
		for (int e = 0; e < nel; e++) {
			for (int m = 0; m < ndof; m++) {
				ind = j*ndof + m;
				LCM[ind][e] = ndof*ECM[j][e] + m;
			}
		}
	}
	printf("LCM\n");
	print_arr((int *)LCM, 2 * ndof, nel);

	//Element local stiffness matrix ke
	//---------------------------------

	for (int k = 0; k < nel; k++) {
		double c = cos(Ang[k]);
		double s = sin(Ang[k]);
		double const_ = CArea[k] * E / L[k];
		double ke[4][4] = { {c*c, c*s, -c*c, -c*s},
					 {c*s, s*s, -c*s, -s*s},
					 {-c*c, -c*s, c*c, c*s},
					 {-c*s, -s*s, c*s, s*s} };

		for (int i = 0; i < 4; i++)
			for (int j = 0; j < 4; j++)
				ke[i][j] = ke[i][j] * const_;

		print_arr((double*)ke, 4, 4);
		// Structure Global stiffenss matrix
		// ---------------------------------

		for (int loop1 = 0; loop1 < nen*ndof; loop1++) {
			int i = LCM[loop1][k];
			for (int loop2 = 0; loop2 < nen*ndof; loop2++) {
				int j = LCM[loop2][k];
				K[i][j] = K[i][j] + ke[loop1][loop2];
			}
		}
	}
	printf("K\n");
	print_arr((double*)K, nn*ndof, nn*ndof);
	//correct upto here


	double Kf[nn*ndof][nn*ndof] = {};
	for (int i = 0; i < nn*ndof; i++){
		for (int j = 0; j < nn*ndof; j++) {
			Kf[i][j] = K[i][j];
		}
    }

	//int Kf[][] = K;

	//Applying Boundary conditions
	//----------------------------

	for (int p = 0; p < sizeof(BC) / sizeof(BC[0]); p++) {
		for (int q = 0; q < tdof; q++) {
			K[BC[p]][ q] = 0;
		}
		K[BC[p]][ BC[p]] = 1;
	}
	

	//Force vector
	//------------

	double f[] = { 0, 0, 0, -200, 0, -100, 0, -100, 0, 0, 0, 0, 0, 0, 0, 0 };

	//Displacement vector
	//-------------------
	const int z = 16;
	double inv[z][z];
	double tt[z][z];
	for (int i = 0; i < z; i++) {
		for (int j = 0; j < z; j++) {
			tt[i][j] = (10 * ((float)rand() / RAND_MAX));
		}
	}
	
	inverse((double*)tt, (double*)inv, z);

	printf("inv\n");
	print_arr((double*)inv, z,z);

	//mat_mul(inv,f);
	//d = K\f

	//Reaction forces
	//---------------

	//mat_mul(Kf,d);
	//fr = Kf*d

	//Axial forces
	//------------
	/*
	for (int e = 0; e < nel; e++) {
		de = d[LCM[:, e]];   //displacement of the current element
		const_ = CArea[e] * E / L[e]; //constant parameter with in the elementt
		p = Ang[e];
		c = cos(e);
		force[e] = const_*{ -c, -s, c, s } *de;
	}*/

	/*
	const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = cudaDeviceReset();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaDeviceReset failed!");
        return 1;
    }*/
	scanf("%d", &ind);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

// Function to get cofactor of A[p][q] in temp[][]. n is current dimension of A[][] 
void getCofactor(double* A, double* temp, int p, int q, int n){
	int i = 0, j = 0;

	// Looping for each element of the matrix 
	for (int row = 0; row < n; row++){
		for (int col = 0; col < n; col++){
			//  Copying into temporary matrix only those element 
			//  which are not in given row and column 
			if (row != p && col != q){
				temp[i*n + j] = A[row*n + col];
				j++;
				// Row is filled, so increase row index and 
				// reset col index 
				if (j == n - 1){
					j = 0;
					i++;
				}
			}
		}
	}
}

/* Recursive function for finding determinant of matrix.
n is current dimension of A[][]. */
double determinant(double* A, int n){
	double D = 0; // Initialize result 

	//  Base case : if matrix contains single element 
	if (n == 1)
		return A[0];

	double* temp =(double*) malloc(sizeof(double)*n*n); // To store cofactors 

	int sign = 1;  // To store sign multiplier 

	// Iterate for each element of first row 
	for (int f = 0; f < n; f++){
		// Getting Cofactor of A[0][f] 
		getCofactor(A, temp, 0, f, n);
		D += sign * A[0*n + f] * determinant(temp, n - 1);
		// terms are to be added with alternate sign 
		sign = -sign;
	}
	free(temp);
	return D;
}

// Function to get adjoint of A[N][N] in adj[N][N]. 
void adjoint(double* A, double* adj, int n){
	if (n == 1){
		adj[0] = 1;
		return;
	}

	// temp is used to store cofactors of A[][] 
	int sign = 1;
	double *temp = (double*) malloc(sizeof(double)*n*n);

	for (int i = 0; i<n; i++){
		for (int j = 0; j<n; j++){
			// Get cofactor of A[i][j] 
			getCofactor(A, temp, i, j, n);

			// sign of adj[j][i] positive if sum of row 
			// and column indexes is even. 
			sign = ((i + j) % 2 == 0) ? 1 : -1;

			// Interchanging rows and columns to get the 
			// transpose of the cofactor matrix 
			adj[j*n + i] = (sign)*(determinant(temp, n - 1));
		}
	}
	free(temp);
}

// Function to calculate and store inverse, returns false if 
// matrix is singular 
bool inverse(double* A, double* inverse, int n){
	// Find determinant of A[][] 
	double det = determinant(A, n);
	printf("%lf\n", det);
	if (det < (double)1e-6 && det > -(double)1e-6){
		//cout << "Singular matrix, can't find its inverse";
		return false;
	}
	// Find adjoint 
	double* adj = (double*)malloc(sizeof(double)*n*n);
	adjoint(A, adj,n);

	// Find Inverse using formula "inverse(A) = adj(A)/det(A)" 
	for (int i = 0; i<n; i++)
		for (int j = 0; j<n; j++)
			inverse[i*n + j] = adj[i*n + j] / double(det);
	free(adj);
	return true;
}

void print_arr(int *arr, int m, int n) {
	int i, j;
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++)
			printf("%d ", *((arr + i*n) + j));
		printf("\n");
	}
}

void print_arr(double *arr, int m, int n) {
	int i, j;
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++)
			printf("%lf ", *((arr + i*n) + j));
		printf("\n");
	}
}